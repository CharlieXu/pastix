#include "hip/hip_runtime.h"
//pas de test sur N


///////////////////////////////////////////////////////////////////////////////////////////////////

// size of work for a thread
#define THR_M ( BLK_M / DIM_X )
#define THR_N ( BLK_N / DIM_Y )

///////////////////////////////////////////////////////////////////////////////////////////////////

#if   (version == trans_nn)
#define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(gemdm_bottom_nn)
#elif (version == trans_nt)
#define TRANS_B
#define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(gemdm_bottom_nt)
#elif (version == trans_nc)
#define TRANS_B
#define CONJ_B
#define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(gemdm_bottom_nc)
#elif (version == trans_tn)
#define TRANS_A
#define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(gemdm_bottom_tn)
#elif (version == trans_tt)
#define TRANS_A
#define TRANS_B
#define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(gemdm_bottom_tt)
#elif (version == trans_tc)
#define TRANS_A
#define TRANS_B
#define CONJ_B
#define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(gemdm_bottom_tc)
#elif (version == trans_cn)
#define TRANS_A
#define CONJ_A
#define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(gemdm_bottom_cn)
#elif (version == trans_ct)
#define TRANS_A
#define CONJ_A
#define TRANS_B
#define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(gemdm_bottom_ct)
#elif (version == trans_cc)
#define TRANS_A
#define CONJ_A
#define TRANS_B
#define CONJ_B
#define kernel_name GENERATE_SM_VERSION_KERNEL_NAME(gemdm_bottom_cc)
#endif

///////////////////////////////////////////////////////////////////////////////////////////////////
extern "C" __global__
void     kernel_name (int M, int N, int K,
		      FloatingPoint_t alpha,
		      const FloatingPoint_t *A, int LDA,
		      const FloatingPoint_t *D, int LDD,
		      const FloatingPoint_t *B, int LDB,
		      FloatingPoint_t beta,
		      FloatingPoint_t       *C, int LDC,
		      int offsetA, int offsetD, int offsetB,
		      int blocknbr, const int *blocktab, int fblocknbr, const int *fblocktab)
{
    int offset[THR_M+1];

    int idx = threadIdx.x;  // thread's m dimension
    int idy = threadIdx.y;  // thread's n dimension

    int idt = DIM_X * idy + idx;    // thread's global number

    int idxA = idt % DIM_XA;    // idx within A
    int idyA = idt / DIM_XA;    // idy within A

    int idxB = idt % DIM_XB;    // idx within B
    int idyB = idt / DIM_XB;    // idy within B

    int blx = M/BLK_M;      // block's m dimension
    int bly = blockIdx.y;   // block's n dimension

    __shared__ FloatingPoint_t sA[BLK_K][BLK_M+1];      // +1 only required if A is transposed
    __shared__ FloatingPoint_t sD[BLK_K];
    __shared__ FloatingPoint_t sB[BLK_N][BLK_K+1];      // +1 always required

    // Registers for the innermost loop
    FloatingPoint_t rC[THR_N][THR_M];
    FloatingPoint_t rA[THR_M];
    FloatingPoint_t rD;
    FloatingPoint_t rB[THR_N];

#ifdef TRANS_A
    const FloatingPoint_t *offs_dA = A + blx*BLK_M*LDA + idyA*LDA+idxA;
#else
    const FloatingPoint_t *offs_dA = A + blx*BLK_M     + idyA*LDA+idxA;
#endif
    const FloatingPoint_t *offs_dD = D + idyA*LDD + idyA;
#ifdef TRANS_B
    const FloatingPoint_t *offs_dB = B + bly*BLK_N     + idyB*LDB+idxB;
#else
    const FloatingPoint_t *offs_dB = B + bly*BLK_N*LDB + idyB*LDB+idxB;
#endif

    int m, n, k, kk;
    int coordm, coordn;
    int coordd;

    // Zero C
    #pragma unroll
    for (n = 0; n < THR_N; n++)
      #pragma unroll
      for (m = 0; m < THR_M; m++)
	rC[n][m] = make_FloatingPoint(0.0, 0.0);

    for (kk = 0; kk < K-BLK_K; kk += BLK_K)
      {
	
	// Load A dev->shmem
#ifdef TRANS_A
    #pragma unroll
    for (n = 0; n < BLK_M; n += DIM_YA){
      #pragma unroll
      for (m = 0; m < BLK_K; m += DIM_XA){
	coordn = n + blx*BLK_M + idyA;
	if(coordn < M)
	  sA[m+idxA][n+idyA] = fetch(A, m, n);
	else sA[m+idxA][n+idyA] = make_FloatingPoint(0.0,0.0);
      }
    }    
    // Load D dev->shmem
    #pragma unroll
    for (n = 0; n < BLK_K; n += DIM_XA)
      {
	    sD[n+idyA] = fetch(D, n, n);

	  }
    #else
    #pragma unroll
    for (n = 0; n < BLK_K; n += DIM_YA){
      #pragma unroll
      for (m = 0; m < BLK_M; m += DIM_XA){
	coordm = m + blx*BLK_M + idxA;
	if(coordm < M)
	  sA[n+idyA][m+idxA] = fetch(A, m, n);
	else sA[n+idyA][m+idxA] = make_FloatingPoint(0.0,0.0);
      }
    }
    
    // Load D dev->shmem
	#pragma unroll
    for (n = 0; n < BLK_K; n += DIM_YA)
      {
	sD[n+idyA] = fetch(D, n, n);
      }
#endif

    // Load B dev->shmem
    #ifdef TRANS_B
    #pragma unroll
    for (n = 0; n < BLK_K; n += DIM_YB){
      #pragma unroll
      for (m = 0; m < BLK_N; m += DIM_XB){
	sB[m+idxB][n+idyB] = fetch(B, m, n);
      }
    }
#else
    #pragma unroll
    for (n = 0; n < BLK_N; n += DIM_YB){
      #pragma unroll
      for (m = 0; m < BLK_K; m += DIM_XB){
	sB[n+idyB][m+idxB] = fetch(B, m, n);
      }
    }
#endif
    
    __syncthreads();

	// Multiply
	#pragma unroll
	for (k = 0; k < BLK_K; k++)
	{
	    rD = sD[k];
	    // Load A shmem->regs
	    #pragma unroll
	    for (m = 0; m < THR_M; m++)
	      {
		rA[m] = mul(sA[k][m*DIM_X+idx],rD);
	      }
	    // Load B shmem->regs
	    #pragma unroll
	    for (n = 0; n < THR_N; n++)
		rB[n] = sB[n*DIM_Y+idy][k];

	    // Compute
	    #pragma unroll
	    for (n = 0; n < THR_N; n++)
		#pragma unroll
		for (m = 0; m < THR_M; m++)
		    #ifdef CONJ_A
		      #ifdef CONJ_B
			fma(conj(rA[m]), conj(rB[n]), rC[n][m]);
		      #else
			fma(conj(rA[m]), rB[n], rC[n][m]);
		      #endif
		    #else
		      #ifdef CONJ_B
			fma(rA[m], conj(rB[n]), rC[n][m]);
		      #else
			fma(rA[m], rB[n], rC[n][m]);
		      #endif
		    #endif
	}

	__syncthreads();


	//maj offset
#ifdef TRANS_A
	offs_dA += BLK_K;
#else
	offs_dA += BLK_K*LDA;
#endif
	offs_dD += BLK_K*LDD + BLK_K;
#ifdef TRANS_B
	offs_dB += BLK_K*LDB;
#else
	offs_dB += BLK_K;
#endif
	
	__syncthreads();
	
      }
    /////////////////////////////////////////////////////////////////////
    
    // Load A dev->shmem
#ifdef TRANS_A
    #pragma unroll
    for (n = 0; n < BLK_M; n += DIM_YA){
      #pragma unroll
      for (m = 0; m < BLK_K; m += DIM_XA){
	coordm = m + idxA + kk;
	coordn = n + blx*BLK_M + idyA;
	if(coordm < K && coordn < M)
	  sA[m+idxA][n+idyA] = fetch(A, m, n);
	else
	  sA[m+idxA][n+idyA] = make_FloatingPoint(0.0,0.0);
      }
    }    
    // Load D dev->shmem
    #pragma unroll
    for (n = 0; n < BLK_K; n += DIM_XA)
      {
	coordd = idyA + kk + n;
	if(coordd<K){
	  sD[n+idyA] = fetch(D, n, n);
	}else 
	   sD[n+idyA] = make_FloatingPoint(1.0,0.0);
	  }
    #else
    #pragma unroll
    for (n = 0; n < BLK_K; n += DIM_YA){
      #pragma unroll
      for (m = 0; m < BLK_M; m += DIM_XA){
	coordm = m + blx*BLK_M + idxA;
	coordn = n + idyA + kk;
	if(coordm < M && coordn < K)
	  sA[n+idyA][m+idxA] = fetch(A, m, n);
	else 
	  sA[n+idyA][m+idxA] = make_FloatingPoint(0.0,0.0);
      }
    }
    
    // Load D dev->shmem
	#pragma unroll
    for (n = 0; n < BLK_K; n += DIM_YA)
      {
	coordd = idyA + kk + n;
	if(coordd<K)
	  sD[n+idyA] = fetch(D, n, n);
	else 
	  sD[n+idyA] = make_FloatingPoint(1.0,0.0);
      }
#endif

    // Load B dev->shmem
    #ifdef TRANS_B
    #pragma unroll
    for (n = 0; n < BLK_K; n += DIM_YB){
      #pragma unroll
      for (m = 0; m < BLK_N; m += DIM_XB){
	coordn = n + idyB + kk;
	if(coordn < K)	
	  sB[m+idxB][n+idyB] = fetch(B, m, n);
	  else 
	    sB[m+idxB][n+idyB] = make_FloatingPoint(0.0,0.0);
      }
    }
#else
    #pragma unroll
    for (n = 0; n < BLK_N; n += DIM_YB){
      #pragma unroll
      for (m = 0; m < BLK_K; m += DIM_XB){
	coordm = m + idxB + kk;
	if(coordm < K)
	  sB[n+idyB][m+idxB] = fetch(B, m, n);
	else 
	  sB[n+idyB][m+idxB] = make_FloatingPoint(0.0,0.0);
      }
    }
#endif
    
    __syncthreads();

	// Multiply
	#pragma unroll
	for (k = 0; k < BLK_K; k++)
	{
	    rD = sD[k];
	    // Load A shmem->regs
	    #pragma unroll
	    for (m = 0; m < THR_M; m++)
	      {
		rA[m] = mul(sA[k][m*DIM_X+idx],rD);
	      }
	    // Load B shmem->regs
	    #pragma unroll
	    for (n = 0; n < THR_N; n++)
		rB[n] = sB[n*DIM_Y+idy][k];

	    // Compute
	    #pragma unroll
	    for (n = 0; n < THR_N; n++)
		#pragma unroll
		for (m = 0; m < THR_M; m++)
		    #ifdef CONJ_A
		      #ifdef CONJ_B
			fma(conj(rA[m]), conj(rB[n]), rC[n][m]);
		      #else
			fma(conj(rA[m]), rB[n], rC[n][m]);
		      #endif
		    #else
		      #ifdef CONJ_B
			fma(rA[m], conj(rB[n]), rC[n][m]);
		      #else
			fma(rA[m], rB[n], rC[n][m]);
		      #endif
		    #endif
	}

	__syncthreads();

	///////////////////////////////////////////////////////////////
	
	{
#define FROWNUM(tab, b) tab[2*b]
#define LROWNUM(tab, b) tab[2*b+1]
#define BLOCKSIZE(tab, b) LROWNUM(tab, b) - FROWNUM(tab, b) + 1
	int blocknum = 0, fblocknum = 0;
	size_t totalblocksize = 0;
	size_t blocksize = BLOCKSIZE(blocktab, blocknum);
	int    rownum;

	offset[0] = 0;
	for (m = 0; m < THR_M; m++) {
	    int coord_dCm = blx*BLK_M + m*DIM_X+idx;

	    if (coord_dCm < M) {

		/*
		 * We should keep blocknum < blocknbr
		 */
		while( totalblocksize + blocksize < coord_dCm + 1)
		{
		    totalblocksize += blocksize;
		    blocknum++;
		    blocksize = BLOCKSIZE(blocktab, blocknum);
		}

		/* Global row index */
		rownum = coord_dCm - totalblocksize + FROWNUM(blocktab, blocknum);

		while (LROWNUM(fblocktab, fblocknum) < rownum) {
		    offset[m] += BLOCKSIZE(fblocktab, fblocknum);
		    fblocknum++;
		}
		offset[m+1] = offset[m];
		offset[m] += rownum - FROWNUM(fblocktab, fblocknum);
	    }
	}
	__syncthreads();
#undef FROWNUM
#undef LROWNUM
    }


    // Store C regs->dev
    #pragma unroll
    for (n = 0; n < THR_N; n++) {
	int coord_dCn = bly*BLK_N + n*DIM_Y+idy;
	#pragma unroll
	for (m = 0; m < THR_M; m++) {
	    int coord_dCm = blx*BLK_M + m*DIM_X+idx;
	    if (coord_dCm < M && coord_dCn < N) {
		int offsC = coord_dCn*LDC + offset[m]; 

		FloatingPoint_t &regC = rC[n][m];
		FloatingPoint_t &memC = C[offsC];

		memC = add(mul(alpha, regC), mul(beta, memC));
	    }
	}
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

#undef TRANS_A
#undef TRANS_B
#undef CONJ_A
#undef CONJ_B

#undef version

#undef THR_M
#undef THR_N

#undef kernel_name
